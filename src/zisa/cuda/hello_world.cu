#include "hip/hip_runtime.h"
#include <zisa/cuda/hello_world.hpp>

#include <stdio.h>

namespace zisa {

// TODO Remove once there is real CUDA code to compile.
__global__ void hello_world_kernel() { printf("hello world.\n"); }

void hello_world() {
  hello_world_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
}
}
