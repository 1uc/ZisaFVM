#include "hip/hip_runtime.h"
#include <zisa/cuda/magma_batched.hpp>

namespace zisa {
namespace cuda {
namespace magma {

namespace internal {
ANY_DEVICE_INLINE
static double *extract_pointer(const array_const_view<double, 2, column_major> &a, int_t k) {
  return const_cast<double *>(&a(0, k));
}

ANY_DEVICE_INLINE
static double *extract_pointer(const array_const_view<double, 3, column_major> &a, int_t k) {
  return const_cast<double *>(&a(0, 0, k));
}

template <int NDIMS>
__global__ void fill_ptrs_kernel(array_view<double *, 1> a_ptrs,
                                 array_const_view<double, NDIMS, column_major> a) {
  auto tid = threadIdx.x;
  int_t i = tid;
  auto n_mat = a_ptrs.shape(0);

  while (i < n_mat) {
    a_ptrs[i] = extract_pointer(a, i);

    i += blockDim.x;
  }
}

template <int NDIMS>
void fill_ptrs_impl(const array_view<double *, 1> &a_ptrs,
                    const array_const_view<double, NDIMS, column_major> &a) {

  // TODO parallelize
  auto n_mat = a_ptrs.shape(0);
  for (int_t i = 0; i < n_mat; ++i) {
    a_ptrs[i] = extract_pointer(a, i);
  }
}

template <int NDIMS>
void fill_ptrs(const array_view<double *, 1> &a_ptrs,
               const array_const_view<double, NDIMS, column_major> &a) {

  auto mem_loc = zisa::memory_location(a_ptrs);
  assert(zisa::memory_location(a) == mem_loc);
  if (mem_loc == device_type::cpu) {
    internal::fill_ptrs_impl(a_ptrs, a);
  } else if (mem_loc == device_type::cuda) {
    internal::fill_ptrs_kernel<<<1024, 1>>>(a_ptrs, a);
    hipDeviceSynchronize();
    ZISA_CHECK_CUDA;
  } else {
    LOG_ERR("Unknown memory location.");
  }
}
}

#define ZISA_FILL_PTRS_DEFN(NDIMS) \
void fill_ptrs(const array_view<double *, 1> &a_ptrs, \
const array_const_view<double, NDIMS, column_major> &a) { \
  internal::fill_ptrs(a_ptrs, a);  \
}

ZISA_FILL_PTRS_DEFN(2);
ZISA_FILL_PTRS_DEFN(3);

#undef ZISA_FILL_PTRS_DECL

}}}